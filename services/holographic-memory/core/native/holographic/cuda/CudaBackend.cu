#include "hip/hip_runtime.h"
#include "CudaBackend.hpp"

#ifdef USE_CUDA_BACKEND
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <stdexcept>
#include <cstring>
#include <cmath>

namespace holo {

// Kernels
__global__ void pad_rows_kernel(const float* __restrict__ in,
                                float* __restrict__ out,
                                uint32_t batch,
                                uint32_t in_cols,
                                uint32_t out_cols) {
    uint32_t b = blockIdx.y;
    uint32_t j = blockIdx.x * blockDim.x + threadIdx.x;
    if (b >= batch || j >= out_cols) return;
    float v = 0.0f;
    if (j < in_cols) v = in[b * in_cols + j];
    out[b * out_cols + j] = v;
}

// Parallel dot/norm reduction with atomics for global accumulation
__global__ void dot_norm_kernel_cuda(const float* __restrict__ a,
                                     const float* __restrict__ b,
                                     float* __restrict__ out_dot,
                                     float* __restrict__ out_n1,
                                     float* __restrict__ out_n2,
                                     uint32_t n) {
    float local_dot = 0.0f;
    float local_n1  = 0.0f;
    float local_n2  = 0.0f;
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t stride = blockDim.x * gridDim.x;
    for (uint32_t i = idx; i < n; i += stride) {
        float x = a[i];
        float y = b[i];
        local_dot += x * y;
        local_n1  += x * x;
        local_n2  += y * y;
    }
    // In-block reduction via warp shuffles
    for (int offset = warpSize/2; offset > 0; offset >>= 1) {
        local_dot += __shfl_down_sync(0xffffffff, local_dot, offset);
        local_n1  += __shfl_down_sync(0xffffffff, local_n1,  offset);
        local_n2  += __shfl_down_sync(0xffffffff, local_n2,  offset);
    }
    // One thread per warp atomically accumulates to global
    if ((threadIdx.x & (warpSize-1)) == 0) {
        atomicAdd(out_dot, local_dot);
        atomicAdd(out_n1,  local_n1);
        atomicAdd(out_n2,  local_n2);
    }
}

__global__ void corr_offset_kernel_cuda(const float* __restrict__ a,
                                        const float* __restrict__ b,
                                        float* __restrict__ out,
                                        uint32_t n,
                                        uint32_t o1,
                                        uint32_t o2) {
    float local = 0.0f;
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t stride = blockDim.x * gridDim.x;
    for (uint32_t i = idx; i < n; i += stride) {
        float x = a[(i + o1) % n];
        float y = b[(i + o2) % n];
        local += x * y;
    }
    for (int offset = warpSize/2; offset > 0; offset >>= 1) {
        local += __shfl_down_sync(0xffffffff, local, offset);
    }
    if ((threadIdx.x & (warpSize-1)) == 0) atomicAdd(out, local);
}

__global__ void magnitude_kernel(const hipfftComplex* __restrict__ in,
                                 float* __restrict__ out,
                                 uint32_t n) {
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    float re = in[idx].x;
    float im = in[idx].y;
    out[idx] = sqrtf(re * re + im * im);
}

CudaBackend::CudaBackend() {}

CudaBackend::~CudaBackend() {
    destroy_plan();
    if (graph_exec_) hipGraphExecDestroy(graph_exec_);
    if (graph_) hipGraphDestroy(graph_);
    if (stream_) hipStreamDestroy(stream_);
    if (d_input_) hipFree(d_input_);
    if (d_fft_in_) hipFree(d_fft_in_);
    if (d_output_) hipFree(d_output_);
    if (h_pinned_) hipHostFree(h_pinned_);
}

bool CudaBackend::available() const noexcept {
    int n=0; if (hipGetDeviceCount(&n) != hipSuccess) return false; return n>0;
}

bool CudaBackend::initialize(const GPUConfig& cfg) {
    config_ = cfg;
    if (!available()) return false;
    int dev = cfg.device_id;
    if (hipSetDevice(dev) != hipSuccess) return false;
    if (!stream_) hipStreamCreate(&stream_);
    create_events();
    initialized_ = true;
    return true;
}

void CudaBackend::destroy_plan() {
    if (fft_plan_) { hipfftDestroy(fft_plan_); fft_plan_ = 0; }
}

void CudaBackend::ensure_buffers(size_t in_bytes, size_t out_bytes, size_t fft_bytes) {
    if (!stream_) hipStreamCreate(&stream_);
    if (in_bytes > in_bytes_) {
        if (d_input_) hipFree(d_input_);
        if (h_pinned_) hipHostFree(h_pinned_);
        hipMalloc(&d_input_, in_bytes);
        hipHostMalloc(&h_pinned_, in_bytes);
        in_bytes_ = in_bytes;
    }
    if (fft_bytes > fft_bytes_) {
        if (d_fft_in_) hipFree(d_fft_in_);
        hipMalloc(&d_fft_in_, fft_bytes);
        fft_bytes_ = fft_bytes;
    }
    if (out_bytes > out_bytes_) {
        if (d_output_) hipFree(d_output_);
        hipMalloc(&d_output_, out_bytes);
        out_bytes_ = out_bytes;
    }
}

void CudaBackend::create_events() {
    if (!start_event_) hipEventCreate(&start_event_);
    if (!end_event_) hipEventCreate(&end_event_);
    if (!fft_start_event_) hipEventCreate(&fft_start_event_);
    if (!fft_end_event_) hipEventCreate(&fft_end_event_);
}

std::vector<std::vector<float>> CudaBackend::batch_encode_fft_ultra(const float* ptr,
                                                                    uint32_t batch,
                                                                    uint32_t data_len,
                                                                    uint32_t pattern_dim) {
    if (!initialized_) initialize({GPUPlatform::CUDA, 0, 0});
    const size_t in_bytes = (size_t)batch * data_len * sizeof(float);
    const size_t fft_bytes = (size_t)batch * pattern_dim * sizeof(float);
    const size_t out_bytes = (size_t)batch * pattern_dim * sizeof(float);
    ensure_buffers(in_bytes, out_bytes, fft_bytes);

    // Rebuild graph if needed
    if (!graph_captured_ || current_pattern_dim_ != pattern_dim) {
        rebuild_graph(batch, data_len, pattern_dim);
    }

    // Host timing
    auto t0 = std::chrono::high_resolution_clock::now();

    // Record overall start event
    hipEventRecord(start_event_, stream_);

    // Copy input to pinned host buffer; graph reads from pinned buffer to device
    std::memcpy(h_pinned_, ptr, in_bytes);

    // Launch captured graph
    hipGraphLaunch(graph_exec_, stream_);
    // Record overall end event
    hipEventRecord(end_event_, stream_);
    hipStreamSynchronize(stream_);

    // Device time (segmented)
    float device_ms = 0.0f, h2d_ms = 0.0f, fft_ms = 0.0f, d2h_ms = 0.0f;
    hipEventElapsedTime(&device_ms, start_event_, end_event_);
    // Note: these events surround captured regions; if captured segments change,
    // adjust placements accordingly.
    hipEventElapsedTime(&fft_ms, fft_start_event_, fft_end_event_);
    // For H2D/D2H, we can place events around memcpy calls pre/post capture in rebuild_graph.
    // Here we report zeros unless events are recorded.
    h2d_ms = 0.0f;
    d2h_ms = 0.0f;

    auto host_ms = std::chrono::duration<double, std::milli>(std::chrono::high_resolution_clock::now()-t0).count();

    // Copy back from device (captured graph already includes D2H into d_output_? we copy now)
    std::vector<float> host_out((size_t)batch * pattern_dim, 0.0f);
    hipMemcpyAsync(host_out.data(), d_output_, out_bytes, hipMemcpyDeviceToHost, stream_);
    hipStreamSynchronize(stream_);

    // Metrics
    metrics_.device_ms = device_ms;
    metrics_.h2d_ms = h2d_ms;
    metrics_.fft_ms = fft_ms;
    metrics_.d2h_ms = d2h_ms;
    metrics_.ops_per_s = (device_ms>0.0f) ? (uint64_t)(batch * 1000.0 / device_ms) : 0;
    metrics_.bandwidth_gbs = (device_ms>0.0f) ? ((double)(in_bytes + out_bytes) / (device_ms/1000.0)) / (1024.0*1024.0*1024.0) : 0.0;

    std::vector<std::vector<float>> out; out.reserve(batch);
    for (uint32_t i=0;i<batch;i++) out.emplace_back(host_out.begin()+ (size_t)i*pattern_dim, host_out.begin()+ (size_t)(i+1)*pattern_dim);
    // Populate device metrics crudely (to be refined with events)
    return out;
}

void CudaBackend::rebuild_graph(uint32_t batch, uint32_t data_len, uint32_t pattern_dim) {
    // Destroy previous
    if (graph_exec_) { hipGraphExecDestroy(graph_exec_); graph_exec_ = nullptr; }
    if (graph_) { hipGraphDestroy(graph_); graph_ = nullptr; }

    // Plan FFT for (batch, pattern_dim)
    destroy_plan();
    int n[1] = {(int)pattern_dim};
    int istride = 1, ostride = 1;
    int inembed[1] = {(int)pattern_dim};
    int onembed[1] = {(int)pattern_dim};
    int idist = (int)pattern_dim, odist = (int)pattern_dim;
    if (hipfftPlanMany(&fft_plan_, 1, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_R2C, (int)batch) != HIPFFT_SUCCESS) {
        throw std::runtime_error("hipfftPlanMany failed in rebuild_graph");
    }
    hipfftSetStream(fft_plan_, stream_);

    // Begin capture
    hipStreamBeginCapture(stream_, hipStreamCaptureModeGlobal);

    // H2D copy from pinned host buffer to device input
    hipEventRecord(h2d_start_event_, stream_);
    hipMemcpyAsync(d_input_, h_pinned_, (size_t)batch * data_len * sizeof(float), hipMemcpyHostToDevice, stream_);
    hipEventRecord(h2d_end_event_, stream_);

    // Pad/truncate to FFT size
    dim3 block(256,1,1);
    dim3 grid((pattern_dim + block.x - 1)/block.x, batch, 1);
    pad_rows_kernel<<<grid, block, 0, stream_>>>((const float*)d_input_, (float*)d_fft_in_, batch, data_len, pattern_dim);

    // FFT
    hipEventRecord(fft_start_event_, stream_);
    hipfftExecR2C(fft_plan_, (hipfftReal*)d_fft_in_, (hipfftComplex*)d_output_);
    hipEventRecord(fft_end_event_, stream_);

    // Magnitude
    const uint32_t total = batch * pattern_dim;
    dim3 mgrid((total + 255)/256, 1, 1);
    magnitude_kernel<<<mgrid, 256, 0, stream_>>>((const hipfftComplex*)d_output_, (float*)d_output_, total);

    // End capture
    hipStreamEndCapture(stream_, &graph_);
    hipGraphInstantiate(&graph_exec_, graph_, nullptr, nullptr, 0);
    graph_captured_ = true;
    current_pattern_dim_ = pattern_dim;
}

std::tuple<float,float,double,float> CudaBackend::analyze_metrics(const float* v1, const float* v2, uint32_t dim) {
    if (!initialized_) initialize({GPUPlatform::CUDA, 0, 0});
    if (!v1 || !v2 || dim == 0) return {0.0f, 0.0f, 0.0, 0.0f};
    // Allocate device buffers for inputs and outputs
    float *d_a=nullptr, *d_b=nullptr, *d_dot=nullptr, *d_n1=nullptr, *d_n2=nullptr;
    float *d_c00=nullptr, *d_c01=nullptr, *d_c20=nullptr, *d_c21=nullptr;
    hipMalloc(&d_a, dim*sizeof(float));
    hipMalloc(&d_b, dim*sizeof(float));
    hipMalloc(&d_dot, sizeof(float)); hipMalloc(&d_n1, sizeof(float)); hipMalloc(&d_n2, sizeof(float));
    hipMalloc(&d_c00, sizeof(float)); hipMalloc(&d_c01, sizeof(float)); hipMalloc(&d_c20, sizeof(float)); hipMalloc(&d_c21, sizeof(float));
    hipMemsetAsync(d_dot, 0, sizeof(float), stream_);
    hipMemsetAsync(d_n1,  0, sizeof(float), stream_);
    hipMemsetAsync(d_n2,  0, sizeof(float), stream_);
    hipMemsetAsync(d_c00, 0, sizeof(float), stream_);
    hipMemsetAsync(d_c01, 0, sizeof(float), stream_);
    hipMemsetAsync(d_c20, 0, sizeof(float), stream_);
    hipMemsetAsync(d_c21, 0, sizeof(float), stream_);
    hipMemcpyAsync(d_a, v1, dim*sizeof(float), hipMemcpyHostToDevice, stream_);
    hipMemcpyAsync(d_b, v2, dim*sizeof(float), hipMemcpyHostToDevice, stream_);

    int threads = 256;
    int blocks = (int)std::min<uint32_t>( (dim + threads - 1)/threads, 1024 );
    dot_norm_kernel_cuda<<<blocks, threads, 0, stream_>>>(d_a, d_b, d_dot, d_n1, d_n2, dim);
    corr_offset_kernel_cuda<<<blocks, threads, 0, stream_>>>(d_a, d_b, d_c00, dim, 0, 0);
    corr_offset_kernel_cuda<<<blocks, threads, 0, stream_>>>(d_a, d_b, d_c01, dim, 0, 1);
    corr_offset_kernel_cuda<<<blocks, threads, 0, stream_>>>(d_a, d_b, d_c20, dim, 2, 0);
    corr_offset_kernel_cuda<<<blocks, threads, 0, stream_>>>(d_a, d_b, d_c21, dim, 2, 1);
    hipStreamSynchronize(stream_);

    float fdot=0.0f, fn1=0.0f, fn2=0.0f, c00=0.0f, c01=0.0f, c20=0.0f, c21=0.0f;
    hipMemcpy(&fdot, d_dot, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&fn1, d_n1, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&fn2, d_n2, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&c00, d_c00, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&c01, d_c01, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&c20, d_c20, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&c21, d_c21, sizeof(float), hipMemcpyDeviceToHost);

    // Clean up
    hipFree(d_a); hipFree(d_b); hipFree(d_dot); hipFree(d_n1); hipFree(d_n2);
    hipFree(d_c00); hipFree(d_c01); hipFree(d_c20); hipFree(d_c21);

    float vis=0.0f, coh=0.0f, ortho = std::fabs(fdot);
    if (fn1 > 0.0f && fn2 > 0.0f) {
        float n1 = std::sqrt(fn1), n2 = std::sqrt(fn2);
        float num = std::fabs(fdot);
        coh = num / (n1 * n2);
        vis = (num*num) / ((n1*n1) * (n2*n2));
    }
    double S = (double)c00/(double)dim + (double)c01/(double)dim + (double)c20/(double)dim - (double)c21/(double)dim;
    double bell_violation = S - 2.0;
    return {vis, coh, bell_violation, ortho};
}

} // namespace holo

#endif
